#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>

void checkCUDAError(const char *msg)
{
 hipError_t err = hipGetLastError();
 if( hipSuccess != err)
 {
  fprintf(stderr, "CUDA Error: %s: %s.\n", msg, hipGetErrorString(err) );
  exit(EXIT_FAILURE); 
 }
}

#define BLOCKSIZE 32

__global__ void dgemm(double* A, double* B, double* C, int N)
{
 size_t myRow = blockIdx.y*blockDim.y+threadIdx.y;
 size_t myCol = blockIdx.x*blockDim.x+threadIdx.x;
  
 if (myRow < N && myCol < N)
  for (size_t i = 0; i < N; i++)
   C[myRow * N + myCol] += A[myRow * N + i] * B[i * N + myCol];
}

int main(int argc, char** argv)
{
  double *A, *B, *C;
  double *dA, *dB, *dC;
  size_t N = 2048;
  
  A = (double*) malloc (sizeof(double)*N*N);
  B = (double*) malloc (sizeof(double)*N*N);
  C = (double*) malloc (sizeof(double)*N*N);

  for (size_t i = 0; i < N; i++)
  for (size_t j = 0; j < N; j++)
  {
   A[i*N + j] = sin(i);
   B[i*N + j] = cos(j);
  }
     
  hipMalloc(&dA, sizeof(double)*N*N); checkCUDAError("Error allocating dA");
  hipMalloc(&dB, sizeof(double)*N*N); checkCUDAError("Error allocating dB");
  hipMalloc(&dC, sizeof(double)*N*N); checkCUDAError("Error allocating dC"); 

  hipMemcpy(dA, A, sizeof(double)*N*N, hipMemcpyHostToDevice); checkCUDAError("Error copying A"); 
  hipMemcpy(dB, B, sizeof(double)*N*N, hipMemcpyHostToDevice); checkCUDAError("Error copying B"); 
  
  auto startTime = std::chrono::system_clock::now();
  
  dim3 threadsPerBlock(BLOCKSIZE, BLOCKSIZE);
  dim3 blocksPerGrid(N/BLOCKSIZE, N/BLOCKSIZE);
  
  dgemm<<<blocksPerGrid,threadsPerBlock>>>(dA, dB, dC, N); checkCUDAError("Failed Kernel Launch"); 
  
  hipMemcpy(C, dC, sizeof(double)*N*N, hipMemcpyDeviceToHost);
  
  hipDeviceSynchronize();
  auto endTime = std::chrono::system_clock::now();

  double checkSum = 0.0;
  for (size_t i = 0; i < N; i++)
  for (size_t j = 0; j < N; j++)
   checkSum += C[i*N + j];
  
  printf("[GPU] Checksum: %f - Elapsed Time: %fs\n", checkSum, std::chrono::duration<double>(endTime-startTime).count());
  
  return 0;
}

 