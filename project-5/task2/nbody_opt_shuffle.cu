/**********************************************************************/
// An unoptimized Naive N-Body solver for Gravity Simulations         //
// G is assumed to be 1.0                                             //
// Course Material for HPCSE-II, Spring 2019, ETH Zurich              //
// Authors: Sergio Martin                                             //
// License: Use if you like, but give us credit.                      //
/**********************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include "string.h"
#include <chrono>


void checkCUDAError(const char *msg);

__global__ void forceKernel(double* xPos, double* yPos, double* zPos, double* mass, double* xFor, double* yFor, double* zFor, size_t N)
{
    size_t m = blockIdx.x*blockDim.x+threadIdx.x;

    for (size_t i = 0; i < N; i++) if (i != m)
    {
        double xDist = xPos[m] - xPos[i];
        double yDist = yPos[m] - yPos[i];
        double zDist = zPos[m] - zPos[i];
        double r     = sqrt(xDist*xDist + yDist*yDist + zDist*zDist);
        xFor[m] += xDist*mass[m]*mass[i] / (r*r*r);
        yFor[m] += yDist*mass[m]*mass[i] / (r*r*r);
        zFor[m] += zDist*mass[m]*mass[i] / (r*r*r);
    }
}


__global__ void OptForceKernel(double* xPos, double* yPos, double* zPos,
                               double* mass,
                               double* xFor, double* yFor, double* zFor,
                               size_t N)
{
    const double EPS= 1e-9; // use to avoid slingshots and division by zero
    size_t threadId= blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId < N)
    {
        size_t laneId= threadIdx.x & 0x1f; // Id within warp
        double xFor_tmp= 0.0; // to accumulate interaction loop results
        double yFor_tmp= 0.0;
        double zFor_tmp= 0.0;

        double xPosRef_reg= xPos[threadId]; // load interaction ref into registers
        double yPosRef_reg= yPos[threadId];
        double zPosRef_reg= zPos[threadId];
        double massRef_reg= mass[threadId];

        for (size_t i= 0; i < N; i+= 32) // warp-tiled interaction with all bodies
        {
            double x= xPos[i + laneId]; // load interaction object
            double y= yPos[i + laneId];
            double z= zPos[i + laneId];
            double m= mass[i + laneId];

            #pragma unroll 32
            for (size_t j= 0; j < 32; ++j) // within warp
            { // use shuffle as it beats shared memory:
                double deltaX= xPosRef_reg - __shfl_sync(0xFFFFFFFF, x, j); 
                double deltaY= yPosRef_reg - __shfl_sync(0xFFFFFFFF, y, j);
                double deltaZ= zPosRef_reg - __shfl_sync(0xFFFFFFFF, z, j);

                double distanceSquared= deltaX * deltaX
                                      + deltaY * deltaY
                                      + deltaZ * deltaZ
                                      + EPS; // avoid slingshots, division by zero
                
                double invDistance= rsqrt(distanceSquared); // use built-in arithmetic
                double invDistanceCubed= invDistance * invDistance * invDistance;
                
                double scalarForce= massRef_reg 
                                  * __shfl_sync(0xFFFFFFFF, m, j) // mass
                                  * invDistanceCubed;
                
                xFor_tmp+= scalarForce * deltaX; // add up xForce vector component
                yFor_tmp+= scalarForce * deltaY; // add up yForce vector component 
                zFor_tmp+= scalarForce * deltaZ; // add up zForce vector component
            }
            __syncthreads();
        }
        xFor[threadId]= xFor_tmp; // write results back
        yFor[threadId]= yFor_tmp;
        zFor[threadId]= zFor_tmp;
    }
}


int main(int argc, char* argv[])
{
    size_t N0 = 80;
    size_t N  = N0*N0*N0;

    // Initializing N-Body Problem

    double* xPos   = (double*) calloc (N, sizeof(double));
    double* yPos   = (double*) calloc (N, sizeof(double));
    double* zPos   = (double*) calloc (N, sizeof(double));
    double* xFor   = (double*) calloc (N, sizeof(double));
    double* yFor   = (double*) calloc (N, sizeof(double));
    double* zFor   = (double*) calloc (N, sizeof(double));
    double* mass   = (double*) calloc (N, sizeof(double));

    size_t current = 0;
    for (size_t i = 0; i < N0; i++)
    for (size_t j = 0; j < N0; j++)
    for (size_t k = 0; k < N0; k++)
    {
        xPos[current] = i;
        yPos[current] = j;
        zPos[current] = k;
        mass[current] = 1.0;
        xFor[current] = 0.0;
        yFor[current] = 0.0;
        zFor[current] = 0.0;
        current++;
    }

    // Allocating and initializing GPU memory

    double* d_xPos; hipMalloc((void **) &d_xPos,  sizeof(double) * N); checkCUDAError("Unable to allocate storage on the device");
    double* d_yPos; hipMalloc((void **) &d_yPos,  sizeof(double) * N); checkCUDAError("Unable to allocate storage on the device");
    double* d_zPos; hipMalloc((void **) &d_zPos,  sizeof(double) * N); checkCUDAError("Unable to allocate storage on the device");
    double* d_xFor; hipMalloc((void **) &d_xFor,  sizeof(double) * N); checkCUDAError("Unable to allocate storage on the device");
    double* d_yFor; hipMalloc((void **) &d_yFor,  sizeof(double) * N); checkCUDAError("Unable to allocate storage on the device");
    double* d_zFor; hipMalloc((void **) &d_zFor,  sizeof(double) * N); checkCUDAError("Unable to allocate storage on the device");
    double* d_mass; hipMalloc((void **) &d_mass,  sizeof(double) * N); checkCUDAError("Unable to allocate storage on the device");

    hipMemcpy(d_xPos, xPos, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Failed Initial Conditions Memcpy");
    hipMemcpy(d_yPos, yPos, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Failed Initial Conditions Memcpy");
    hipMemcpy(d_zPos, zPos, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Failed Initial Conditions Memcpy");
    hipMemcpy(d_xFor, xFor, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Failed Initial Conditions Memcpy");
    hipMemcpy(d_yFor, yFor, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Failed Initial Conditions Memcpy");
    hipMemcpy(d_zFor, zFor, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Failed Initial Conditions Memcpy");
    hipMemcpy(d_mass, mass, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Failed Initial Conditions Memcpy");

    // Calculating Kernel Geometry
    size_t threadsPerBlock  = 512;
    size_t blocksPerGrid    = ceil(double (((double)N) / ((double)threadsPerBlock)));

    // Running Force-calculation kernel
    auto startTime = std::chrono::system_clock::now();
    OptForceKernel<<<blocksPerGrid, threadsPerBlock>>>(d_xPos, d_yPos, d_zPos,
                                                       d_mass,
                                                       d_xFor, d_yFor, d_zFor, N);
    checkCUDAError("Failed Force Kernel");
    hipDeviceSynchronize();
    auto endTime = std::chrono::system_clock::now();

    hipMemcpy(xFor, d_xFor, sizeof(double) * N, hipMemcpyDeviceToHost); checkCUDAError("Failed Final Conditions Memcpy");
    hipMemcpy(yFor, d_yFor, sizeof(double) * N, hipMemcpyDeviceToHost); checkCUDAError("Failed Final Conditions Memcpy");
    hipMemcpy(zFor, d_zFor, sizeof(double) * N, hipMemcpyDeviceToHost); checkCUDAError("Failed Final Conditions Memcpy");

    hipFree(d_xPos);
    hipFree(d_yPos);
    hipFree(d_zPos);
    hipFree(d_xFor);
    hipFree(d_yFor);
    hipFree(d_zFor);
    hipFree(d_mass);

    double netForce = 0.0;
    double absForce = 0.0;
    for (size_t i = 0; i < N; i++) netForce += xFor[i] + yFor[i] + zFor[i];
    for (size_t i = 0; i < N; i++) absForce += abs(xFor[i] + yFor[i] + zFor[i]);

    printf("     Net Force: %.6f\n", netForce);
    printf("Absolute Force: %.6f\n", absForce);

    if (isfinite(netForce) == false)      { printf("Verification Failed: Net force is not a finite value!\n"); exit(-1); }
    if (fabs(netForce) > 0.00001)         { printf("Verification Failed: Force equilibrium not conserved!\n"); exit(-1); }
    if (isfinite(absForce) == false)      { printf("Verification Failed: Absolute Force is not a finite value!\n"); exit(-1); }

    printf("Time: %.8fs\n", std::chrono::duration<double>(endTime-startTime).count());
    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "CUDA Error: %s: %s.\n", msg, hipGetErrorString(err) );
        exit(EXIT_FAILURE);
    }
}
